#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>

void generateRandomMatrix(int rows, int cols, int *matrix)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            matrix[i * cols + j] = rand() % 10;
        }
    }
}

void printMatrix(int rows, int cols, int *matrix)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%d\t", matrix[i * cols + j]);
        }
        printf("\n");
    }
    // printf("\n");
}

__global__ void matrixMultiplicationGPU(int rowsA, int colsA, int colsB, int *matrixA, int *matrixB, int *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rowsA && j < colsB)
    {
        int sum = 0;
        for (int k = 0; k < colsA; k++)
        {
            sum += matrixA[i * colsA + k] * matrixB[k * colsB + j];
        }
        result[i * colsB + j] = sum;
    }
}

int main()
{

    int rows, cols;
    printf("Type number of rows: ");
    std::cin >> rows;
    printf("Type number of columns: ");
    std::cin >> cols;

    int size = rows * cols;
    size_t bytes = size * sizeof(int);

    // Allocate memory for matrices on CPU
    int *matrixA_CPU = (int *)malloc(bytes);
    int *matrixB_CPU = (int *)malloc(bytes);
    int *result_CPU = (int *)malloc(bytes);

    // Allocate memory for matrices on GPU
    int *matrixA_GPU;
    int *matrixB_GPU;
    int *result_GPU;
    hipMalloc((void **)&matrixA_GPU, bytes);
    hipMalloc((void **)&matrixB_GPU, bytes);
    hipMalloc((void **)&result_GPU, bytes);

    // Generate random matrices
    generateRandomMatrix(rows, cols, matrixA_CPU);
    generateRandomMatrix(rows, cols, matrixB_CPU);

    // Copy matrices from CPU to GPU
    hipMemcpy(matrixA_GPU, matrixA_CPU, bytes, hipMemcpyHostToDevice);
    hipMemcpy(matrixB_GPU, matrixB_CPU, bytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    clock_t start_timer, end_timer;

    start_timer = clock();
    matrixMultiplicationGPU<<<numBlocks, threadsPerBlock>>>(rows, cols, cols, matrixA_GPU, matrixB_GPU, result_GPU);
    end_timer = clock();
    double timer = ((double)end_timer - start_timer) / CLOCKS_PER_SEC;

    hipMemcpy(result_CPU, result_GPU, bytes, hipMemcpyDeviceToHost);
    printf("\nMatrix Multiplication : \n");
    // printMatrix(rows, cols, result_CPU);
    printf("Time taken : %fs \n", timer);

    // Free memory
    free(matrixA_CPU);
    free(matrixB_CPU);
    free(result_CPU);
    hipFree(matrixA_GPU);
    hipFree(matrixB_GPU);
    hipFree(result_GPU);
}